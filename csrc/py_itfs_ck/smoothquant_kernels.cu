#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include "py_itfs_common.h"

#include "moe_smoothquant.hpp"


void moe_smoothquant_fwd(torch::Tensor &out,      // [topk * tokens, hidden_size]
                         torch::Tensor &input,    // [tokens, hidden_size]
                         torch::Tensor &x_scale,  // [experts, hidden_size]
                         torch::Tensor &topk_ids, // [tokens, topk]
                         torch::Tensor &y_scale)  // [topk * tokens,  1]
{
    auto dtype = input.dtype();
    TORCH_CHECK(dtype == torch::kFloat16 || dtype == torch::kBFloat16,
                "ck smoothquant only support fp16 and bf16 data type");

    std::string dtype_str = torchDTypeToStr(input.dtype());
    int n = input.size(-1);
    int m = input.numel() / n;
    int experts = x_scale.size(0);
    int topk = topk_ids.size(1);
    int stride = n;
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    moe_smoothquant({
                        dtype_str // input  dtype
                    },
                    {input.data_ptr(),    // [tokens, hidden_size], input, fp16/bf16
                     x_scale.data_ptr(),  // [experts, hidden_size], input, columnwise scale, fp32
                     topk_ids.data_ptr(), // [tokens, topk]

                     y_scale.data_ptr(), // [topk * tokens,  1], output, rowwise quant scale
                     out.data_ptr(),     // [topk * tokens, hidden_size], output
                     m, n, experts, topk, stride, stride},
                    {stream});
}
